#include <hip/hip_runtime.h>
#include "check.h"
#include "initialize.h"

static hipDeviceProp_t
current_device_prop()
{
  int device;
  CUDA_CHECK(hipGetDevice(&device));

  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, device));

  return prop;
}

void
ispm_initialize()
{
  hipDeviceProp_t p = current_device_prop();
  int arch = p.major * 10 + p.minor;
  if (arch != CUDA_ARCH)
  {
    fprintf(stderr, "CUDA device compute capability mismatch: "
            "compiled for sm_%d, got sm_%d\n", CUDA_ARCH, arch);
    abort();
  }
  CUDA_CHECK(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_CHECK(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
}

DeviceIDString
get_device_string()
{
  hipDeviceProp_t p = current_device_prop();

  DeviceIDString s;
  snprintf(s.str, sizeof(s.str), "%04x:%02x:%02x (%s)",
           p.pciDomainID, p.pciBusID, p.pciDeviceID, p.name);

  return s;
}
